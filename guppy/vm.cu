#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>

/*
#define BYTECODE_OP static inline __device__

BYTECODE_OP void load_slice() {

}

BYTECODE_OP void add(void* a, void *b) {

}
*/
double Now() {
    timespec tp;
    clock_gettime(CLOCK_MONOTONIC, &tp);
    return tp.tv_sec + 1e-9 * tp.tv_nsec;
}

#define TIMEOP(op)\
{\
  double st = Now();\
  op;\
  double ed = Now();\
  fprintf(stderr, "%s finished in %.f seconds.\n", #op, end - start);\
}


enum OP_CODE {
  LOAD_SLICE, STORE_SLICE,    // load slice of global arrays into shared vector
  LOAD_SCALAR, STORE_SCALAR, // distribute scalar across elements of shared vector
  ADD, SUB, MUL, DIV,        // arithmetic between shared vectors
  BAD
};

struct Op {
  Op() : code(BAD), x(0), y(0), z(0) {}
  Op(OP_CODE code, int x, int y, int z) : code(code), x(x), y(y), z(z)  {}

  OP_CODE code; 
  int x, y, z;
};

struct Program {
	std::vector<Op> _ops;
	Op* _gpu_ptr;


	Program& Add(int x, int y, int z) {
		_ops.push_back(Op(ADD, x, y, z));
		return *this;
	}
	Program& LoadSlice(int src, int dst) {
		_ops.push_back(Op(LOAD_SLICE, src, dst, 0));
		return *this;
	}
	Program& StoreSlice(int src, int dst) {
		_ops.push_back(Op(LOAD_SLICE, src, dst, 0));
		return *this;
	}

	int size() {
      return _ops.size();
	}

	int nbytes () {
	  return sizeof(Op) * this->size();
	}

	Op* host_ptr() {
	  return &_ops[0];
	}
	Op* to_gpu() {
	  if (_gpu_ptr) {
		  return _gpu_ptr;
	  }
	  hipMalloc(&_gpu_ptr, this->nbytes());
	  hipMemcpy(_gpu_ptr, this->host_ptr(), this->nbytes(), hipMemcpyHostToDevice);
	  return _gpu_ptr;
	}

	Program() : _gpu_ptr(NULL) {}
	~Program () {
	  if (_gpu_ptr) {
	    hipFree(_gpu_ptr);
	  }
	}
};


struct Vec {
  int _n;
  int _nbytes;
  float* _host_data;
  float* _gpu_data;
  bool _host_dirty;
  bool _gpu_dirty;

  void init(int n) {
    _n = n;
    _nbytes = sizeof(float) * n;
    _host_data = new float[n];
	hipMalloc(&_gpu_data, this->_nbytes);
    _host_dirty = false;
    _gpu_dirty = true;
  }

  Vec(int n) {
    this->init(n);
  }

  Vec (int n, float fill_value) {
    this->init(n);
    for (int i = 0; i < n; ++i) {
      _host_data[i] = fill_value;
    }
  }

  float* get_gpu_data() {
     if (_gpu_dirty) { this->copy_to_gpu(); }
     _host_dirty = true;
     _gpu_dirty = false;
     return _gpu_data;
  }

  float* get_host_data() {
	  if (_host_dirty) { this->copy_to_host();}
	  _gpu_dirty = true;
	  _host_dirty = false;
	  return _host_data;
  }

  void copy_to_host() {
	  hipMemcpy(this->_host_data, this->_gpu_data, this->_nbytes, hipMemcpyDeviceToHost);
  }


  void copy_to_gpu() {
	  hipMemcpy(this->_gpu_data, this->_host_data, this->_nbytes, hipMemcpyHostToDevice);
  }

  ~Vec() {
	  hipFree(this->_gpu_data);
	  delete[] this->_host_data;
  }

};

#define REGISTER_WIDTH 128
#define NUM_REGISTERS 16

__global__ void run(
		Op* program, int n_ops,
		float** values, int n_args,
		float* constants, int n_consts) {
  int startIdx = blockIdx.x * blockDim.x; 
  // int stopIdx = startIdx + blockDim.x;
  __shared__ float registers[NUM_REGISTERS][REGISTER_WIDTH];

  for (int pc = 0; pc < n_ops; ++pc) {
    Op op = program[pc];
    switch (op.code) {
    case LOAD_SLICE: {
      float* dst = registers[op.y] + threadIdx.x;
      float* src = values[op.x] + startIdx + threadIdx.x;
      *dst = *src;
    }
    break;

    case STORE_SLICE: {
      float* dst = values[op.y] + startIdx + threadIdx.x;
      float* src = registers[op.x] + threadIdx.x;
      *dst = *src;
    }
    break;

    case LOAD_SCALAR: {

    }
    break;

    case STORE_SCALAR: {

    }
    break;

	case ADD: {
	    float* x = registers[op.x] + threadIdx.x; //+ startIdx + threadIdx.x;
	    float* y = registers[op.y] + threadIdx.x; //+ startIdx + threadIdx.x;
	    float* z = values[op.z] + threadIdx.x; //+ startIdx + threadIdx.x;
        *z = *x + *y;
      }
	break;
    }  
  }
}

#define THREADS_PER_BLOCK 512

int main(int argc, const char** argv) { 
  int N = 400 * THREADS_PER_BLOCK;
  if (argc > 1) {
    N = strtol(argv[1], NULL, 10);
  }
    
  Vec a(N, 1.0);
  Vec b(N, 2.0);
  Vec c(N);
  
  const int n_values = 3;
  float* h_values[n_values];
  h_values[0]= a.get_gpu_data();
  h_values[1] = b.get_gpu_data();
  h_values[2] = c.get_gpu_data();

  float** d_values;
  hipMalloc(&d_values, sizeof(float*) * n_values);
  hipMemcpy(d_values, h_values, sizeof(float*) * n_values, hipMemcpyHostToDevice);


  Program h_program;

  h_program.
    LoadSlice(0,0).
    LoadSlice(1,1).
    Add(0,1,2).
    StoreSlice(2,2);

  double st = Now();
  run<<<N / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(
		  h_program.to_gpu(), h_program.size(),
		  d_values, n_values,
		  0, 0);
  hipDeviceSynchronize();
  double ed = Now();
  fprintf(stderr, "%.5f seconds\n", ed -st);

  float* ad = a.get_host_data();
  printf("%f %f %f\n", ad[0], ad[1], ad[2]);
  float* cd = c.get_host_data();
  printf("%f %f %f\n", cd[0], cd[1], cd[2]);
  return 0; 
}
